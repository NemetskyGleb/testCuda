﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

hipError_t addWithCuda();

__global__ void addKernel()
{
    printf("%d, %d", threadIdx.x, threadIdx.y);
}

int main()
{
    hipError_t cudaStatus = addWithCuda();

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda()
{
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        std::exit(-1);
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, 12>>>();

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        std::exit(-1);
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        std::exit(-1);
    }

    return cudaStatus;
}
